#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>

#include <opencv2/opencv.hpp>

#define CHECK(call) \
{                                                       \
    const hipError_t error = call;                     \
    if (error != hipSuccess)                           \
    {                                                   \
        printf("Error: %s,%d, ", __FILE__, __LINE__);   \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));\
        exit(1);\
    }\
}

struct RGB
{
    float   R;
    float   G;
    float   B;

    __device__  __host__ void Set(float r, float g, float b)
    {
        R = r;
        G = g;
        B = b;
    }

};

__device__ void SetRgb(RGB *rgb, float r, float g, float b)
{
    rgb->R = r;
    rgb->G = g;
    rgb->B = b;
}


__device__ RGB ConvertHSVtoRGB(float H, float S, float V)
{
  float C = V * S;
  float HH = H / 60;
  float X = C * (1.0f - fabs(fmod(HH, 2.0f) - 1.0f));

  RGB rgb_temp;

  if (HH >= 0 && HH <= 1)
  {
      SetRgb(&rgb_temp,C,X,0);
  }
  else if (HH >= 1 && HH <= 2)
  {
      SetRgb(&rgb_temp,X,C,0);
  }
  else if (HH >= 2 && HH <= 3)
  {
      SetRgb(&rgb_temp,0,C,X);
  }
  else if (HH >= 3 && HH <= 4)
  {
      SetRgb(&rgb_temp,0,X,C);
  } 
  else if (HH >= 4 && HH <= 5)
  {
      SetRgb(&rgb_temp,X,0,C);
  }
  else if (HH >= 5 && HH < 6)
  {
      SetRgb(&rgb_temp,C,0,X);
  }

  float m = V - C;

  RGB rgb;

  SetRgb(&rgb, rgb_temp.R + m, rgb_temp.G + m, rgb_temp.B + m);

  return rgb;
}


__device__ float function_table(float in)//-407
{
    //5,                         -407.0, 0.1,//8752
    //                            3345.0, 70.0,//5000
    //                            7344.0, 135.0,//1000
    //                            8345.0, 150.0,//0
    //                            8752.0, 250.0,

   float input[5] = {-407.0f,3345.0f, 7344.0f,8345.0f,8752.0f};
   float output[5] = {0.1f,70.0f,135.0f,150.0f,250.0f};
   

   float out;
    
    if (in <= input[0])
    {
        out = output[0];
    }
    else if (in > input[0] && in < input[4])
    {
        for (int ii = 0; ii < 4; ii++)
        {
            if (in > input[ii] && in <= input[ii+1])
            {
                out = output[ii] + (in - input[ii]) / (input[ii + 1] - input[ii]) * (output[ii + 1] - output[ii]);
                break;
            }
        }
    }
    else
    {
        out = output[4];
    }


    return out;
}

__global__ void ElvationToRGB(short *elev, RGB* rgbValues)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId >= 10800 * 4)
        return;

    short mb =   elev[threadId];

    if (mb > 8752)
        mb = 8752;
    else if (mb < -407)
        mb = -407;

    mb =  8752 - 407 - mb;

    float H = function_table(mb);

    RGB rgb  = ConvertHSVtoRGB(H,1,1);

    rgbValues[threadId] = rgb;
}

int main(int argc, char **argv)
{  

    int totalColumns = 10800 * 4;

    int columns = 10800;
    int rows[4] = {4800, 6000, 6000, 4800};

    int totalRows = rows[0] + rows[1] + rows[2] + rows[3];

    const char* fileNames[4][4] = { "all10//a10g", "all10//b10g","all10//c10g","all10//d10g",
                            "all10//e10g", "all10//f10g","all10//g10g","all10//h10g",
                            "all10//i10g", "all10//j10g","all10//k10g","all10//l10g",
                            "all10//m10g", "all10//n10g","all10//o10g","all10//p10g"};

    //std::ofstream frame;
    //frame.open("c10_43200_21600.ppm", std::ios::trunc);

    //frame << "P3\n" << totalColumns<< " " << 4800 + 6000 + 6000 + 4800<< "\n255\n";

    short *memblock = new short[totalColumns * totalRows];
    
    int rowCount = 0;

    for (int ii = 0; ii < 4; ii++)
    {
        std::ifstream file1(fileNames[ii][0], std::ios::in | std::ios::binary); 
        std::ifstream file2(fileNames[ii][1], std::ios::in | std::ios::binary);  
        std::ifstream file3(fileNames[ii][2], std::ios::in | std::ios::binary);  
        std::ifstream file4(fileNames[ii][3], std::ios::in | std::ios::binary); 

        file1.seekg(0, std::ios::beg); 
        file2.seekg(0, std::ios::beg); 
        file3.seekg(0, std::ios::beg); 
        file4.seekg(0, std::ios::beg);  

        if (file1.is_open() && file2.is_open() && file3.is_open() && file4.is_open()/* && frame.is_open()*/)
        {

            for (int jj = 0; jj < rows[ii]; jj++)
            {
                file1.read((char*)&memblock[totalColumns * rowCount + 0], columns * sizeof(short));
                file2.read((char*)&memblock[totalColumns * rowCount + columns], columns * sizeof(short));
                file3.read((char*)&memblock[totalColumns * rowCount + columns * 2], columns * sizeof(short));
                file4.read((char*)&memblock[totalColumns * rowCount + columns * 3], columns * sizeof(short));

                //std::cout << "major: " << ii << ", row: " << jj << std::endl;
                rowCount++;
            }

            file1.close();
            file2.close();
            file3.close();
            file4.close(); 
        } 
        else
        {
            std::cout << "batch: " << ii << " one of the files unable to open" << std::endl;
            std::cin.get();
            return 0;
        } 
    }

    std::cout << "Finished reading elevations" << std::endl;

    cv::Size globeSize(totalColumns, totalRows);
    cv::Mat globeImage(globeSize, CV_8UC3);

    std::cout << "Created globeImage" << std::endl;

    short *d_memblock;
    CHECK(hipMalloc((short **)&d_memblock, totalColumns * sizeof(short)));
    RGB *d_rgbValues;
    CHECK(hipMalloc((RGB **)&d_rgbValues, totalColumns * sizeof(RGB)));
    RGB  *h_rgbValues = (RGB*)malloc(sizeof(RGB) * totalColumns);


    clock_t startTimeCuda = clock();

    rowCount = 0;

    while (rowCount < totalRows)
    {

      CHECK(hipMemcpy(d_memblock, &memblock[rowCount * totalColumns], sizeof(short) * totalColumns, hipMemcpyHostToDevice));

      int blockSize = 1024;
      int numBlocks = (totalColumns + blockSize - 1) / blockSize;

      ElvationToRGB <<<numBlocks, blockSize>>>(d_memblock,d_rgbValues);   

      hipError_t err = hipGetLastError();

      char any;
       
      if (err != hipSuccess) 
      {
          printf("Error: %s\n", hipGetErrorString(err));
          std::cin >> any;
      }

      CHECK(hipDeviceSynchronize()); 

      CHECK(hipMemcpy(h_rgbValues, d_rgbValues, sizeof(RGB) * totalColumns, hipMemcpyDeviceToHost));



      uchar *ptr = globeImage.ptr(rowCount);
      for (int col = 0; col < globeImage.cols; col++)
      {
          uchar * uc_pixel = ptr;
          uc_pixel[0] = (int)(h_rgbValues[col].B * 255);
          uc_pixel[1] = (int)(h_rgbValues[col].G * 255);
          uc_pixel[2] = (int)(h_rgbValues[col].R * 255);           
          ptr += 3;
      }


      rowCount++;
    }

    std::cout << "Finished computing rgb values in " << double( clock() - startTimeCuda ) / (double)CLOCKS_PER_SEC<< " seconds." << std::endl;

    std::cout << "Started globe.png write" << std::endl;

    clock_t startTime = clock();

    cv::imwrite( "globe.png", globeImage );

    std::cout << "Finished globe.png in " << double( clock() - startTime ) / (double)CLOCKS_PER_SEC<< " seconds." << std::endl;


 

    delete [] memblock;
    hipFree(d_memblock);
    hipFree(d_rgbValues);
    free(h_rgbValues);

    hipDeviceReset();

}




        
